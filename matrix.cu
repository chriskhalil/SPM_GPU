
#include "matrix.h"

#include <assert.h>
#include <cstdlib>
#include <stdio.h>

COOMatrix* createEmptyCOOMatrix(unsigned int numRows, unsigned int numCols, unsigned int capacity) {
    COOMatrix *cooMatrix = (COOMatrix *) malloc(sizeof(COOMatrix));
    cooMatrix->numRows = numRows;
    cooMatrix->numCols = numCols;
    cooMatrix->numNonzeros = 0;
    cooMatrix->capacity = capacity;
    cooMatrix->rowIdxs = (unsigned int*) calloc(1, capacity*sizeof(unsigned int));
    cooMatrix->colIdxs = (unsigned int*) malloc(capacity*sizeof(unsigned int));
    cooMatrix->values = (float*) malloc(capacity*sizeof(float));
    return cooMatrix;
}

void freeCOOMatrix(COOMatrix* cooMatrix) {
    free(cooMatrix->rowIdxs);
    free(cooMatrix->colIdxs);
    free(cooMatrix->values);
    free(cooMatrix);
}

void merge(unsigned int* key1, unsigned int *key2, float *data, int left, int mid, int right, unsigned int* key1Aux, unsigned int* key2Aux, float* dataAux) {

    int size1 = mid - left + 1;
    int size2 = right - mid;

    unsigned int *key1Left  = key1Aux; key1Aux += size1;
    unsigned int *key1Right = key1Aux; key1Aux += size2;
    unsigned int *key2Left  = key2Aux; key2Aux += size1;
    unsigned int *key2Right = key2Aux; key2Aux += size2;
    float *dataLeft  = dataAux; dataAux += size1;
    float *dataRight = dataAux; dataAux += size2;

    for(int i = 0; i < size1; ++i) {
        key1Left[i] = key1[left + i];
        key2Left[i] = key2[left + i];
        dataLeft[i] = data[left + i];
    }
    for(int j = 0; j < size2; ++j) {
        key1Right[j] = key1[mid + 1 + j];
        key2Right[j] = key2[mid + 1 + j];
        dataRight[j] = data[mid + 1 + j];
    }

    int i = 0, j = 0, k = left;
    while (i < size1 && j < size2) {
        if (key1Left[i] < key1Right[j] || key1Left[i] == key1Right[j] && key2Left[i] <= key2Right[j]) {
            key1[k] = key1Left[i];
            key2[k] = key2Left[i];
            data[k] = dataLeft[i];
            ++i;
            ++k;
        } else {
            key1[k] = key1Right[j];
            key2[k] = key2Right[j];
            data[k] = dataRight[j];
            ++j;
            ++k;
        }
    }
    while (i < size1) {
        key1[k] = key1Left[i];
        key2[k] = key2Left[i];
        data[k] = dataLeft[i];
        ++i;
        ++k;
    }
    while (j < size2) {
        key1[k] = key1Right[j];
        key2[k] = key2Right[j];
        data[k] = dataRight[j];
        ++j;
        ++k;
    }

}

void mergeSort(unsigned int *key1, unsigned int *key2, float *data, int start, int end, unsigned int* key1Aux, unsigned int* key2Aux, float* dataAux) {
    if (start < end) {
        int mid = start + (end - start) / 2;
        mergeSort(key1, key2, data, start, mid, key1Aux, key2Aux, dataAux);
        mergeSort(key1, key2, data, mid + 1, end, key1Aux, key2Aux, dataAux);
        merge(key1, key2, data, start, mid, end, key1Aux, key2Aux, dataAux);
    }
}

void sortCOOMatrix(COOMatrix* cooMatrix) {
    unsigned int* key1Aux = (unsigned int*) malloc(cooMatrix->numNonzeros*sizeof(unsigned int));
    unsigned int* key2Aux = (unsigned int*) malloc(cooMatrix->numNonzeros*sizeof(unsigned int));
    float* dataAux = (float*) malloc(cooMatrix->numNonzeros*sizeof(float));
    mergeSort(cooMatrix->rowIdxs, cooMatrix->colIdxs, cooMatrix->values, 0, cooMatrix->numNonzeros - 1, key1Aux, key2Aux, dataAux);
    free(key1Aux);
    free(key2Aux);
    free(dataAux);
}

COOMatrix* createEmptyCOOMatrixOnGPU(unsigned int numRows, unsigned int numCols, unsigned int capacity) {

    COOMatrix cooMatrixShadow;
    cooMatrixShadow.numRows = numRows;
    cooMatrixShadow.numCols = numCols;
    cooMatrixShadow.numNonzeros = 0;
    cooMatrixShadow.capacity = capacity;
    hipMalloc((void**) &cooMatrixShadow.rowIdxs, capacity*sizeof(unsigned int));
    hipMalloc((void**) &cooMatrixShadow.colIdxs, capacity*sizeof(unsigned int));
    hipMalloc((void**) &cooMatrixShadow.values, capacity*sizeof(float));

    COOMatrix* cooMatrix;
    hipMalloc((void**) &cooMatrix, sizeof(COOMatrix));
    hipMemcpy(cooMatrix, &cooMatrixShadow, sizeof(COOMatrix), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    return cooMatrix;

}

void clearCOOMatrixOnGPU(COOMatrix* cooMatrix) {
    COOMatrix cooMatrixShadow;
    hipMemcpy(&cooMatrixShadow, cooMatrix, sizeof(COOMatrix), hipMemcpyDeviceToHost);
    hipMemset(cooMatrixShadow.rowIdxs, 0, cooMatrixShadow.numNonzeros*sizeof(unsigned int));
    hipMemset(cooMatrixShadow.colIdxs, 0, cooMatrixShadow.numNonzeros*sizeof(unsigned int));
    hipMemset(cooMatrixShadow.values, 0, cooMatrixShadow.numNonzeros*sizeof(unsigned int));
    hipMemset(&cooMatrix->numNonzeros, 0, sizeof(unsigned int));
}

void freeCOOMatrixOnGPU(COOMatrix* cooMatrix) {
    COOMatrix cooMatrixShadow;
    hipMemcpy(&cooMatrixShadow, cooMatrix, sizeof(COOMatrix), hipMemcpyDeviceToHost);
    hipFree(cooMatrixShadow.rowIdxs);
    hipFree(cooMatrixShadow.colIdxs);
    hipFree(cooMatrixShadow.values);
    hipFree(cooMatrix);
}

void copyCOOMatrixFromGPU(COOMatrix* cooMatrix_d, COOMatrix* cooMatrix_h) {
    COOMatrix cooMatrixShadow;
    hipMemcpy(&cooMatrixShadow, cooMatrix_d, sizeof(COOMatrix), hipMemcpyDeviceToHost);
    assert(cooMatrix_h->numRows == cooMatrixShadow.numRows);
    assert(cooMatrix_h->numCols == cooMatrixShadow.numCols);
    assert(cooMatrix_h->capacity >= cooMatrixShadow.numNonzeros);
    cooMatrix_h->numNonzeros = cooMatrixShadow.numNonzeros;
    hipMemcpy(cooMatrix_h->rowIdxs, cooMatrixShadow.rowIdxs, cooMatrixShadow.numNonzeros*sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(cooMatrix_h->colIdxs, cooMatrixShadow.colIdxs, cooMatrixShadow.numNonzeros*sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(cooMatrix_h->values, cooMatrixShadow.values, cooMatrixShadow.numNonzeros*sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
}

COOMatrix* readCOOMatrix(const char* fileName) {

    COOMatrix* cooMatrix = (COOMatrix*) malloc(sizeof(COOMatrix));

    // Initialize fields
    FILE* fp = fopen(fileName, "r");
    int x = 1;
    x |= fscanf(fp, "%u", &cooMatrix->numRows);
    cooMatrix->numCols = cooMatrix->numRows;
    x |= fscanf(fp, "%u", &cooMatrix->numNonzeros);
    cooMatrix->rowIdxs = (unsigned int*) malloc(cooMatrix->numNonzeros*sizeof(unsigned int));
    cooMatrix->colIdxs = (unsigned int*) malloc(cooMatrix->numNonzeros*sizeof(unsigned int));
    cooMatrix->values = (float*) malloc(cooMatrix->numNonzeros*sizeof(float));

    // Read the nonzeros
    for(unsigned int i = 0; i < cooMatrix->numNonzeros; ++i) {
        x |= fscanf(fp, "%u", &cooMatrix->rowIdxs[i]);
        x |= fscanf(fp, "%u", &cooMatrix->colIdxs[i]);
        cooMatrix->values[i] = 0.5 + ((float)rand())/RAND_MAX;
    }

    return cooMatrix;

}

CSRMatrix* coo2csr(COOMatrix* cooMatrix) {

    CSRMatrix* csrMatrix = (CSRMatrix*) malloc(sizeof(CSRMatrix));;

    // Initialize fields
    csrMatrix->numRows = cooMatrix->numRows;
    csrMatrix->numCols = cooMatrix->numCols;
    csrMatrix->numNonzeros = cooMatrix->numNonzeros;
    csrMatrix->rowPtrs = (unsigned int*) malloc((csrMatrix->numRows + 1)*sizeof(unsigned int));
    csrMatrix->colIdxs = (unsigned int*) malloc(csrMatrix->numNonzeros*sizeof(unsigned int));
    csrMatrix->values = (float*) malloc(csrMatrix->numNonzeros*sizeof(float));

    // Histogram rows
    memset(csrMatrix->rowPtrs, 0, (csrMatrix->numRows + 1)*sizeof(unsigned int));
    for(unsigned int i = 0; i < cooMatrix->numNonzeros; ++i) {
        unsigned int row = cooMatrix->rowIdxs[i];
        csrMatrix->rowPtrs[row]++;
    }

    // Prefix sum rowPtrs
    unsigned int sumBeforeNextRow = 0;
    for(unsigned int row = 0; row < csrMatrix->numRows; ++row) {
        unsigned int sumBeforeRow = sumBeforeNextRow;
        sumBeforeNextRow += csrMatrix->rowPtrs[row];
        csrMatrix->rowPtrs[row] = sumBeforeRow;
    }
    csrMatrix->rowPtrs[csrMatrix->numRows] = sumBeforeNextRow;

    // Bin the nonzeros
    for(unsigned int i = 0; i < cooMatrix->numNonzeros; ++i) {
        unsigned int row = cooMatrix->rowIdxs[i];
        unsigned int j = csrMatrix->rowPtrs[row]++;
        csrMatrix->colIdxs[j] = cooMatrix->colIdxs[i];
        csrMatrix->values[j] = cooMatrix->values[i];
    }

    // Restore rowPtrs
    for(unsigned int row = csrMatrix->numRows - 1; row > 0; --row) {
        csrMatrix->rowPtrs[row] = csrMatrix->rowPtrs[row - 1];
    }
    csrMatrix->rowPtrs[0] = 0;

    return csrMatrix;

}

CSRMatrix* createCSRMatrixFromFile(const char* fileName) {
    COOMatrix* cooMatrix = readCOOMatrix(fileName);
    CSRMatrix* csrMatrix = coo2csr(cooMatrix);
    freeCOOMatrix(cooMatrix);
    return csrMatrix;
}

void freeCSRMatrix(CSRMatrix* csrMatrix) {
    free(csrMatrix->rowPtrs);
    free(csrMatrix->colIdxs);
    free(csrMatrix->values);
    free(csrMatrix);
}

CSRMatrix* createEmptyCSRMatrixOnGPU(unsigned int numRows, unsigned int numCols, unsigned int numNonzeros) {

    CSRMatrix csrMatrixShadow;
    csrMatrixShadow.numRows = numRows;
    csrMatrixShadow.numCols = numCols;
    csrMatrixShadow.numNonzeros = numNonzeros;
    hipMalloc((void**) &csrMatrixShadow.rowPtrs, (numRows + 1)*sizeof(unsigned int));
    hipMalloc((void**) &csrMatrixShadow.colIdxs, numNonzeros*sizeof(unsigned int));
    hipMalloc((void**) &csrMatrixShadow.values, numNonzeros*sizeof(float));

    CSRMatrix* csrMatrix;
    hipMalloc((void**) &csrMatrix, sizeof(CSRMatrix));
    hipMemcpy(csrMatrix, &csrMatrixShadow, sizeof(CSRMatrix), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    return csrMatrix;

}

void freeCSRMatrixOnGPU(CSRMatrix* csrMatrix) {
    CSRMatrix csrMatrixShadow;
    hipMemcpy(&csrMatrixShadow, csrMatrix, sizeof(CSRMatrix), hipMemcpyDeviceToHost);
    hipFree(csrMatrixShadow.rowPtrs);
    hipFree(csrMatrixShadow.colIdxs);
    hipFree(csrMatrixShadow.values);
    hipFree(csrMatrix);
}

void copyCSRMatrixToGPU(CSRMatrix* csrMatrix_h, CSRMatrix* csrMatrix_d) {
    CSRMatrix csrMatrixShadow;
    hipMemcpy(&csrMatrixShadow, csrMatrix_d, sizeof(CSRMatrix), hipMemcpyDeviceToHost);
    assert(csrMatrixShadow.numRows == csrMatrix_h->numRows);
    assert(csrMatrixShadow.numCols == csrMatrix_h->numCols);
    assert(csrMatrixShadow.numNonzeros == csrMatrix_h->numNonzeros);
    hipMemcpy(csrMatrixShadow.rowPtrs, csrMatrix_h->rowPtrs, (csrMatrix_h->numRows + 1)*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(csrMatrixShadow.colIdxs, csrMatrix_h->colIdxs, csrMatrix_h->numNonzeros*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(csrMatrixShadow.values, csrMatrix_h->values, csrMatrix_h->numNonzeros*sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
}

