#include "Utility.cuh"

GpuConfig& GpuConfig::GetInstance() {
	//Thanks Meyer: check book 
	static GpuConfig _instance;
	return _instance;
}

GpuConfig::GpuConfig() {
	
	int num_of_devices{ 0 };
	auto errata{ hipGetDeviceCount(&num_of_devices) };

	if (errata == hipSuccess)
	{
		/// <summary>
		/// We will assume that we are only using one and only one gpu
		/// for this project.
		/// </summary>
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop,0);

		_name=prop.name;
		_id=0;
		_cuda_version=prop.major;
		_total_memory=prop.totalGlobalMem;
		_shared_mem_per_block=prop.sharedMemPerBlock;
		_total_const_mem=prop.totalConstMem;
		_registers_per_block=prop.regsPerBlock;
		_warp_size=prop.warpSize;
		_max_threads_per_block=prop.maxThreadsPerBlock;

		__copy(prop.maxThreadsDim, _max_threads_dim, 3);
		__copy(prop.maxGridSize, _max_grid_size, 3);

		_multiprocessor_count=prop.multiProcessorCount;
	}
}

string	 GpuConfig::Name() const {
	return _name;
}
int_32	 GpuConfig::Id() const {
	return _id;
}
int_32	 GpuConfig::CudaVersion() const {
	return _cuda_version;
}
size_t	 GpuConfig::TotalDeviceMem() const {

	return _total_memory;

}
size_t	 GpuConfig::TotalConstMem()	 const {
	return _total_const_mem;
}
size_t	 GpuConfig::SharedBlockMem() const {
	
	return _shared_mem_per_block;
}
int_32	 GpuConfig::RegistersPerBlock() const {
	return _registers_per_block;
}
int_32	 GpuConfig::WarpSize()	const {
	return _warp_size;
}
int_32	 GpuConfig::MaxThreadPerBlock() const {
	return _max_threads_per_block;
}
const int*    GpuConfig::MaxThreadDim(){
	return _max_threads_dim;
}
const int*    GpuConfig::MaxGridSize(){
	return _max_grid_size;
}
int_32	 GpuConfig::MultiprocessorCount() const {
	return _multiprocessor_count;

}
void	GpuConfig::PrintInfo(){
	cout << "Gpu Name:" << Name() << "\n";
	cout << "Cuda Version:" << CudaVersion() << "\n";
	cout << "Total Global Memory:"<< TotalDeviceMem() << " B  ~"<<GetApproxSize(TotalDeviceMem(),UnitMeasure::GB)<<" GB\n";
	cout << "Total Const  Memory:" << TotalConstMem() << " B  ~"<<GetApproxSize(TotalConstMem(),UnitMeasure::KB)<<" KB\n";
	cout << "Shared Memory Per Block:"  << SharedBlockMem() << " B  ~"<<GetApproxSize(SharedBlockMem() ,UnitMeasure::KB)<<" KB\n";
	cout << "Registers Per Block:" << RegistersPerBlock() << "\n";
	cout << "Warp Size:" << WarpSize() << "\n";
	const auto* tmp = MaxThreadDim();
	cout << "Max Thread Dim: x:" <<tmp[0]<<" y:"<<tmp[1]<<" z:"<<tmp[2] << "\n";
	tmp = MaxGridSize();
    cout << "Max Grid Dim: x:" << tmp[0] << " y:" << tmp[1] << " z:" << tmp[2] << "\n";
	cout << "Multi Processors Count (SM):" << MultiprocessorCount() << "\n";



}

float GpuConfig::extract_size(size_t bytes, const UnitMeasure sz) noexcept {
    //simple function with low decimal precision

    size_t _whole{ bytes / static_cast<size_t>(sz) };   
    bytes -= (static_cast<size_t>(sz) * _whole);
    switch (sz)
    {
    case UnitMeasure::TB:
        //go for gb in digit
        return _whole + (bytes / static_cast<size_t>(UnitMeasure::GB)) * 0.001;
    case UnitMeasure::GB:
        //go for mB in digit
        return _whole + (bytes / static_cast<size_t>(UnitMeasure::MB)) * 0.001;
    case UnitMeasure::MB:
        return _whole + (bytes / static_cast<size_t>(UnitMeasure::KB)) * 0.001;
    case UnitMeasure::KB:
        return _whole + (bytes / static_cast<size_t>(UnitMeasure::B)) * 0.001;
    default:
        //unexpected UnitMeasure size
        return -1;
    } 
}

float GpuConfig::GetApproxSize(size_t mem,const UnitMeasure ms){
	return extract_size(mem,ms);
}

